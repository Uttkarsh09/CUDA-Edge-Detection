#include "hip/hip_runtime.h"
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "helper_timer.h"

#define KERNEL_SIZE    5

__global__ void gaussianBlurKernel(unsigned char *input_image, unsigned char *output_image, int width, int height, float *kernel, int kernelSize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        float blurPixel = 0.0f;
        int kernelRadius = kernelSize / 2;

        for (int i = -kernelRadius; i <= kernelRadius; i++)
        {
            for (int j = -kernelRadius; j <= kernelRadius; j++)
            {
                int xOffset = x + i;
                int yOffset = y + j;

                if (xOffset >= 0 && xOffset < width && yOffset >= 0 && yOffset < height)
                {
                    int inputIndex = yOffset * width + xOffset;
                    int kernelIndex = (i + kernelRadius) * kernelSize + (j + kernelRadius);
                    blurPixel = blurPixel + static_cast<float>(input_image[inputIndex]) * kernel[kernelIndex];
                }
            }
        }

        output_image[y * width + x] = static_cast<unsigned char>(blurPixel);
    }
}

void gaussianBlurGPU(cv::Mat *inputImage, cv::Mat *outputImage)
{
    // Variable Declarations
    hipError_t result;
    StopWatchInterface *timer = nullptr;
    unsigned char *deviceInput = nullptr, *deviceOutput = nullptr;
    float *hostKernel = nullptr, *deviceKernel = nullptr;
    float kernelSum = 0.0f;
    float sigma = 1.0f;

    int imageWidth = inputImage->cols;
    int imageHeight = inputImage->rows;
    int imageSize = imageHeight * imageWidth * sizeof(unsigned char);
    
    // Create Gaussian Kernel
    hostKernel = new float[KERNEL_SIZE * KERNEL_SIZE];
    int kernelRadius = KERNEL_SIZE / 2;

    for (int i = -kernelRadius; i <= kernelRadius; i++) 
    {
        for (int j = -kernelRadius; j <= kernelRadius; j++)
        {
            int index = (i + kernelRadius) * kernelRadius + (j + kernelRadius);
            hostKernel[index] = exp(-(i * i + j + j) / (2.0f * sigma * sigma));
            kernelSum = kernelSum + hostKernel[index];
        }
    }

    for (int i = 0; i < KERNEL_SIZE * KERNEL_SIZE; i++)
    {
        hostKernel[i] = hostKernel[i] / kernelSum;
    }

    sdkCreateTimer(&timer);

    result = hipMalloc((void **)&deviceInput, imageSize);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Input Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceOutput, imageSize);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Output Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceKernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(float));
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Device Kernel ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceInput, inputImage->data, imageSize, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Input Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceKernel, hostKernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Device Kernel ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Kernel Configuration
    dim3 dimBlock(16, 16);
    dim3 dimGrid((imageWidth + dimBlock.x - 1) / dimBlock.x, (imageHeight + dimBlock.y - 1) / dimBlock.y);

    sdkStartTimer(&timer);
    gaussianBlurKernel<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, imageWidth, imageHeight, deviceKernel, KERNEL_SIZE);
    sdkStopTimer(&timer);

    std::cout << std::endl << "Time for Gaussian using CUDA : " << sdkGetTimerValue(&timer) << " ms" << std::endl;

    result = hipMemcpy(outputImage->data, deviceOutput, imageSize, hipMemcpyDeviceToHost);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Output Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    if (deviceKernel)
    {
        hipFree(deviceKernel);
        deviceKernel = nullptr;
    }

    if (deviceOutput)
    {
        hipFree(deviceOutput);
        deviceOutput = nullptr;
    }

    if (deviceInput)
    {
        hipFree(deviceInput);
        deviceInput = nullptr;
    }

    if (hostKernel)
    {
        delete[] hostKernel;
        hostKernel = nullptr;
    }

    if (timer)
    {
        sdkDeleteTimer(&timer);
        timer = nullptr;
    }
}

int main(void)
{
    cv::Mat input_image, output_image;

    input_image = cv::imread("Images\\Input\\pikachu.jpg", cv::IMREAD_GRAYSCALE);
    output_image = input_image.clone();

    gaussianBlurGPU(&input_image, &output_image);

    output_image.convertTo(output_image, CV_8UC1);

    cv::imwrite("Images\\Output\\Gaussian-CUDA.jpg", output_image);

    input_image.release();
    output_image.release();
    
    exit(EXIT_SUCCESS);
}
