#include "hip/hip_runtime.h"
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "helper_timer.h"

#define BLOCK_SIZE            32
#define GRID_SIZE             128
#define SOBEL_KERNEL_SIZE     5
#define GAUSSIAN_KERNEL_SIZE  3

float gaussianBlurTime, sobelFilterTime;

__global__ void gaussianBlurKernel(unsigned char *input_image, unsigned char *output_image, int width, int height, float *kernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        float blurPixel = 0.0f;
        int kernelRadius = GAUSSIAN_KERNEL_SIZE / 2;

        for (int i = -kernelRadius; i <= kernelRadius; i++)
        {
            for (int j = -kernelRadius; j <= kernelRadius; j++)
            {
                int xOffset = x + i;
                int yOffset = y + j;

                if (xOffset >= 0 && xOffset < width && yOffset >= 0 && yOffset < height)
                {
                    int inputIndex = yOffset * width + xOffset;
                    int kernelIndex = (i + kernelRadius) * GAUSSIAN_KERNEL_SIZE + (j + kernelRadius);
                    blurPixel = blurPixel + static_cast<float>(input_image[inputIndex]) * kernel[kernelIndex];
                }
            }
        }

        output_image[y * width + x] = static_cast<unsigned char>(blurPixel);
    }
}

__global__ void sobelFilterKernel(unsigned char *input_image, unsigned char *output_image, unsigned int image_width, unsigned int image_height)
{
    int sobel_x[SOBEL_KERNEL_SIZE][SOBEL_KERNEL_SIZE] = {
        { -1, 0, 1 },
        { -2, 0, 2 },
        { -1, 0, 1 }
    };

    int sobel_y[SOBEL_KERNEL_SIZE][SOBEL_KERNEL_SIZE] = {
        { -1, -2, -1 },
        { 0, 0, 0 },
        { 1, 2, 1 }
    };

    int num_rows = blockIdx.x * blockDim.x + threadIdx.x;
    int num_columns = blockIdx.y * blockDim.y + threadIdx.y;

    int index = (num_rows * image_width) + num_columns;

    if ((num_columns < (image_width - 1)) && (num_rows < (image_height - 1)))
    {
        float gradient_x =  (input_image[index] * sobel_x[0][0]) + (input_image[index + 1] * sobel_x[0][1]) + (input_image[index + 2] * sobel_x[0][2]) +
                            (input_image[index] * sobel_x[1][0]) + (input_image[index + 1] * sobel_x[1][1]) + (input_image[index + 2] * sobel_x[1][2]) +
                            (input_image[index] * sobel_x[2][0]) + (input_image[index + 1] * sobel_x[2][1]) + (input_image[index + 2] * sobel_x[2][2]);

        float gradient_y =  (input_image[index] * sobel_y[0][0]) + (input_image[index + 1] * sobel_y[0][1]) + (input_image[index + 2] * sobel_y[0][2]) +
                            (input_image[index] * sobel_y[1][0]) + (input_image[index + 1] * sobel_y[1][1]) + (input_image[index + 2] * sobel_y[1][2]) +
                            (input_image[index] * sobel_y[2][0]) + (input_image[index + 1] * sobel_y[2][1]) + (input_image[index + 2] * sobel_y[2][2]);

        float gradient = sqrtf(gradient_x * gradient_x + gradient_y * gradient_y);

        if (gradient > 255)
            gradient = 255;

        if (gradient < 0)
            gradient = 0;

        __syncthreads();

        output_image[index] = gradient;
    }
}

void gaussianBlurGPU(cv::Mat *inputImage, cv::Mat *outputImage)
{
    // Variable Declarations
    hipError_t result;
    StopWatchInterface *timer = nullptr;
    unsigned char *deviceInput = nullptr, *deviceOutput = nullptr;
    float *hostKernel = nullptr, *deviceKernel = nullptr;
    float kernelSum = 0.0f;
    float sigma = 1.0f;

    int imageWidth = inputImage->cols;
    int imageHeight = inputImage->rows;
    int imageSize = imageHeight * imageWidth * sizeof(unsigned char);
    
    // Create Gaussian Kernel
    hostKernel = new float[GAUSSIAN_KERNEL_SIZE * GAUSSIAN_KERNEL_SIZE];
    int kernelRadius = GAUSSIAN_KERNEL_SIZE / 2;

    for (int i = -kernelRadius; i <= kernelRadius; i++) 
    {
        for (int j = -kernelRadius; j <= kernelRadius; j++)
        {
            int index = (i + kernelRadius) * kernelRadius + (j + kernelRadius);
            hostKernel[index] = exp(-(i * i + j + j) / (2.0f * sigma * sigma));
            kernelSum = kernelSum + hostKernel[index];
        }
    }

    for (int i = 0; i < GAUSSIAN_KERNEL_SIZE * GAUSSIAN_KERNEL_SIZE; i++)
    {
        hostKernel[i] = hostKernel[i] / kernelSum;
    }

    sdkCreateTimer(&timer);

    result = hipMalloc((void **)&deviceInput, imageSize);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Input Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceOutput, imageSize);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Output Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceKernel, GAUSSIAN_KERNEL_SIZE * GAUSSIAN_KERNEL_SIZE * sizeof(float));
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Device Kernel ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceInput, inputImage->data, imageSize, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Input Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceKernel, hostKernel, GAUSSIAN_KERNEL_SIZE * GAUSSIAN_KERNEL_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Device Kernel ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Kernel Configuration
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(imageHeight, imageWidth);

    sdkStartTimer(&timer);
    gaussianBlurKernel<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, imageWidth, imageHeight, deviceKernel);
    sobelFilterKernel<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, inputImage->cols, inputImage->rows);
    sdkStopTimer(&timer);

    std::cout << std::endl << "Time for Sobel Operator using CUDA (GPU) : " << sdkGetTimerValue(&timer) << " ms" << std::endl;

    result = hipMemcpy(outputImage->data, deviceOutput, imageSize, hipMemcpyDeviceToHost);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Output Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    if (deviceKernel)
    {
        hipFree(deviceKernel);
        deviceKernel = nullptr;
    }

    if (deviceOutput)
    {
        hipFree(deviceOutput);
        deviceOutput = nullptr;
    }

    if (deviceInput)
    {
        hipFree(deviceInput);
        deviceInput = nullptr;
    }

    if (hostKernel)
    {
        delete[] hostKernel;
        hostKernel = nullptr;
    }

    if (timer)
    {
        sdkDeleteTimer(&timer);
        timer = nullptr;
    }
}

void sobelGPU(cv::Mat *inputImage, cv::Mat *outputImage)
{
    unsigned char *deviceInput = nullptr, *deviceOutput = nullptr;
    hipError_t result;
    StopWatchInterface *timer = nullptr;

    int imageSize = inputImage->rows * inputImage->cols * sizeof(unsigned char);

    sdkCreateTimer(&timer);

    result = hipMalloc((void **)&deviceInput, imageSize);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Input Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceOutput, imageSize);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Output Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceInput, inputImage->data, imageSize, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Input Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    const dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 numBlocks(inputImage->cols, inputImage->rows);

    sdkStartTimer(&timer);
    sobelFilterKernel<<<numBlocks, threadsPerBlock>>>(deviceInput, deviceOutput, inputImage->cols, inputImage->rows);
    sdkStopTimer(&timer);
    sobelFilterTime = sdkGetTimerValue(&timer);

    result = hipMemcpy(outputImage->data, deviceOutput, imageSize, hipMemcpyDeviceToHost);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Output Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    if (deviceOutput)
    {
        hipFree(deviceOutput);
        deviceOutput = nullptr;
    }

    if (deviceInput)
    {
        hipFree(deviceInput);
        deviceInput = nullptr;
    }

    if (timer)
    {
        sdkDeleteTimer(&timer);
        timer = nullptr;
    }
}

int main(void)
{
    cv::Mat input_image, output_image;

    input_image = cv::imread("Images\\Input\\win1.jpg", cv::IMREAD_GRAYSCALE);
    output_image = input_image.clone();

    gaussianBlurGPU(&input_image, &output_image);
    // sobelGPU(&input_image, &output_image);

    output_image.convertTo(output_image, CV_8UC1);

    cv::imwrite("Images\\Output\\Sobel-CUDA.jpg", output_image);

    input_image.release();
    output_image.release();
    
    exit(EXIT_SUCCESS);
}
