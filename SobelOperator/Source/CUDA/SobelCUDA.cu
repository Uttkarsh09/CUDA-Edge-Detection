#include "hip/hip_runtime.h"
#include "../../Headers/CUDA/SobelCUDA.cuh"

cv::Mat input_image, output_image;
cv::String input_file, output_file;
StopWatchInterface *timer = nullptr;
unsigned char *deviceInput = nullptr, *deviceOutput = nullptr;
float *hostKernel = nullptr, *deviceKernel = nullptr;

__global__ void gaussianBlurKernel(unsigned char *input_image, unsigned char *output_image, int width, int height, float *kernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        float blurPixel = 0.0f;
        int kernelRadius = GAUSSIAN_KERNEL_SIZE / 2;

        for (int i = -kernelRadius; i <= kernelRadius; i++)
        {
            for (int j = -kernelRadius; j <= kernelRadius; j++)
            {
                int xOffset = x + i;
                int yOffset = y + j;

                if (xOffset >= 0 && xOffset < width && yOffset >= 0 && yOffset < height)
                {
                    int inputIndex = yOffset * width + xOffset;
                    int kernelIndex = (i + kernelRadius) * GAUSSIAN_KERNEL_SIZE + (j + kernelRadius);
                    blurPixel = blurPixel + static_cast<float>(input_image[inputIndex]) * kernel[kernelIndex];
                }
            }
        }

        output_image[y * width + x] = static_cast<unsigned char>(blurPixel);
    }
}

__global__ void sobelFilterKernel(unsigned char *input_image, unsigned char *output_image, unsigned int image_width, unsigned int image_height)
{
    int sobel_x[SOBEL_KERNEL_SIZE][SOBEL_KERNEL_SIZE] = {
        { -1, 0, 1 },
        { -2, 0, 2 },
        { -1, 0, 1 }
    };

    int sobel_y[SOBEL_KERNEL_SIZE][SOBEL_KERNEL_SIZE] = {
        { -1, -2, -1 },
        { 0, 0, 0 },
        { 1, 2, 1 }
    };

    int num_rows = blockIdx.x * blockDim.x + threadIdx.x;
    int num_columns = blockIdx.y * blockDim.y + threadIdx.y;

    int index = (num_rows * image_width) + num_columns;

    if ((num_columns < (image_width - 1)) && (num_rows < (image_height - 1)))
    {
        float gradient_x =  (input_image[index] * sobel_x[0][0]) + (input_image[index + 1] * sobel_x[0][1]) + (input_image[index + 2] * sobel_x[0][2]) +
                            (input_image[index] * sobel_x[1][0]) + (input_image[index + 1] * sobel_x[1][1]) + (input_image[index + 2] * sobel_x[1][2]) +
                            (input_image[index] * sobel_x[2][0]) + (input_image[index + 1] * sobel_x[2][1]) + (input_image[index + 2] * sobel_x[2][2]);

        float gradient_y =  (input_image[index] * sobel_y[0][0]) + (input_image[index + 1] * sobel_y[0][1]) + (input_image[index + 2] * sobel_y[0][2]) +
                            (input_image[index] * sobel_y[1][0]) + (input_image[index + 1] * sobel_y[1][1]) + (input_image[index + 2] * sobel_y[1][2]) +
                            (input_image[index] * sobel_y[2][0]) + (input_image[index + 1] * sobel_y[2][1]) + (input_image[index + 2] * sobel_y[2][2]);

        float gradient = sqrtf(gradient_x * gradient_x + gradient_y * gradient_y);

        if (gradient > 255)
            gradient = 255;

        if (gradient < 0)
            gradient = 0;

        __syncthreads();

        output_image[index] = gradient;
    }
}

void runSobelOperator(cv::Mat *inputImage, cv::Mat *outputImage)
{
    // Variable Declarations
    hipError_t result;
    float kernelSum = 0.0f;
    float sigma = 1.0f;

    int imageWidth = inputImage->cols;
    int imageHeight = inputImage->rows;
    int imageSize = imageHeight * imageWidth * sizeof(unsigned char);
    
    // Create Gaussian Kernel
    hostKernel = new float[GAUSSIAN_KERNEL_SIZE * GAUSSIAN_KERNEL_SIZE];
    int kernelRadius = GAUSSIAN_KERNEL_SIZE / 2;

    for (int i = -kernelRadius; i <= kernelRadius; i++) 
    {
        for (int j = -kernelRadius; j <= kernelRadius; j++)
        {
            int index = (i + kernelRadius) * kernelRadius + (j + kernelRadius);
            hostKernel[index] = exp(-(i * i + j + j) / (2.0f * sigma * sigma));
            kernelSum = kernelSum + hostKernel[index];
        }
    }

    for (int i = 0; i < GAUSSIAN_KERNEL_SIZE * GAUSSIAN_KERNEL_SIZE; i++)
    {
        hostKernel[i] = hostKernel[i] / kernelSum;
    }

    sdkCreateTimer(&timer);

    result = hipMalloc((void **)&deviceInput, imageSize);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Input Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceOutput, imageSize);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Output Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceKernel, GAUSSIAN_KERNEL_SIZE * GAUSSIAN_KERNEL_SIZE * sizeof(float));
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMalloc() Failed For Device Kernel ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceInput, inputImage->data, imageSize, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Input Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceKernel, hostKernel, GAUSSIAN_KERNEL_SIZE * GAUSSIAN_KERNEL_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Device Kernel ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Kernel Configuration
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(imageHeight, imageWidth);

    sdkStartTimer(&timer);
    gaussianBlurKernel<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, imageWidth, imageHeight, deviceKernel);
    sobelFilterKernel<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, inputImage->cols, inputImage->rows);
    sdkStopTimer(&timer);

    result = hipMemcpy(outputImage->data, deviceOutput, imageSize, hipMemcpyDeviceToHost);
    if (result != hipSuccess)
    {
        std::cerr << std::endl << "hipMemcpy() Failed For Output Image ... Exiting !!!" << std::endl;
        exit(EXIT_FAILURE);
    }
}

void sobelCUDA(int image_number)
{
    switch(image_number)
    {
        case 1:
            input_file = "Images\\Input\\img1.jpg";
            output_file = "Images\\Output\\Sobel-CUDA-1.jpg";
        break;
        case 2:
            input_file = "Images\\Input\\img2.jpg";
            output_file = "Images\\Output\\Sobel-CUDA-2.jpg";
        break;
        case 3:
            input_file = "Images\\Input\\img3.jpg";
            output_file = "Images\\Output\\Sobel-CUDA-3.jpg";
        break;
        case 4:
            input_file = "Images\\Input\\img4.jpg";
            output_file = "Images\\Output\\Sobel-CUDA-4.jpg";
        break;
        case 5:
            input_file = "Images\\Input\\img5.jpg";
            output_file = "Images\\Output\\Sobel-CUDA-5.jpg";
        break;
        default:
            std::cerr << std::endl << "Error ... Please Enter Valid Number ... Exiting !!!" << std::endl;
            cleanup();
            exit(EXIT_FAILURE);
        break;
    }

    input_image = cv::imread(input_file, cv::IMREAD_GRAYSCALE);
    output_image = input_image.clone();

    runSobelOperator(&input_image, &output_image);

    std::cout << std::endl << "Time for Sobel Operator using CUDA (GPU) : " << sdkGetTimerValue(&timer) << " ms" << std::endl;

    output_image.convertTo(output_image, CV_8UC1);

    cv::imwrite(output_file, output_image);

    cleanup();
}

void cleanup(void)
{
    if (deviceKernel)
    {
        hipFree(deviceKernel);
        deviceKernel = nullptr;
    }

    if (deviceOutput)
    {
        hipFree(deviceOutput);
        deviceOutput = nullptr;
    }

    if (deviceInput)
    {
        hipFree(deviceInput);
        deviceInput = nullptr;
    }

    if (hostKernel)
    {
        delete[] hostKernel;
        hostKernel = nullptr;
    }

    if (timer)
    {
        sdkDeleteTimer(&timer);
        timer = nullptr;
    }

    output_image.release();
    input_image.release();
}
